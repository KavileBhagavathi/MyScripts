
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <time.h>
double getTimeStamp()
    {
        struct timespec ts;
        clock_gettime(CLOCK_MONOTONIC, &ts);
        return (double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9;
    }

void initoncpu(double *a_src, size_t size)
    {
        for(size_t i=0; i<size; i++){
           a_src[i] = 1.0;
        }
    }

void checkOnCPU(double *a_dest, size_t numberofelements) 
    {
        for (size_t i = 0; i < numberofelements; ++i)
        {
        assert(2.0 == a_dest[i]);
        }
        printf("Assertion success!\n");
    }

__global__ void cudakernel(double *d_src, double *d_dest, size_t numberofelements)
    {
        size_t start = blockIdx.x * blockDim.x + threadIdx.x;
        size_t stride = gridDim.x * blockDim.x;
        for (size_t i = start; i < numberofelements; i=i+stride)
        {
            d_dest[i] = d_src[i] + 1.0;
        }
    }

__global__ void cudakernelwarmup(double *d_src, double *d_dest, size_t numberofelements)
    {
        size_t start = blockIdx.x * blockDim.x + threadIdx.x;
        size_t stride = gridDim.x * blockDim.x;
        for (size_t i = start; i < numberofelements; i=i+stride)
        {
            d_dest[i] = 1.0;
        }
    }

int main(int argc, char **argv0)
    {
    double kernel_wct_start,kernel_wct_end,hosttodev_start,hosttodev_end,devtohost_start,devtohost_end;
    size_t numberofelements = atoi(argv0[1]);
    size_t kernel_reps = atoi(argv0[2]);
    size_t cuda_blocks = atoi(argv0[3]);
    size_t cuda_threads_per_block = atoi(argv0[4]);
    FILE *fptr;
    fptr = fopen("cudavararrop.txt","a");
    if(cuda_threads_per_block==1)
    {
        fprintf(fptr,"Number_of_elements,Kernel_reps,Cuda_blocks,Threads_per_block,T_hosttodev,T_exec,T_devtohost,Bw_HosttoDev,Bw_DevtoHost,Bw_execution\n");
    }
    size_t size = sizeof(double)*numberofelements;
    //allocate host arrays
    double *a_src, *a_dest;
    if (hipHostMalloc(&a_src, size, hipHostMallocDefault) != hipSuccess || hipHostMalloc(&a_dest, size, hipHostMallocDefault) != hipSuccess) 
        {
            printf("Failed to allocate host memory.\n");
            return 1;
        }
    else
        {
            printf("Allocated host memory.\n");
        }
    // allocate _device_ arrays
    double *d_src, *d_dest;
    if (hipMalloc(&d_src, size) != hipSuccess || hipMalloc(&d_dest, size) != hipSuccess) 
        {
            printf("Failed to allocate device memory.\n");
            return 1;
        }
    else
        {
            printf("Allocated device memory.\n");
        }
    //initialize data on CPU
    initoncpu(a_src,numberofelements);
    hosttodev_start = getTimeStamp();
    hipMemcpy(d_src, a_src, size, hipMemcpyHostToDevice);
    hosttodev_end = getTimeStamp();
        
    auto numBlocks = atoi(argv0[3]);
    auto numThreadsPerBlock = atoi(argv0[4]);
    
    cudakernelwarmup<<<numBlocks, numThreadsPerBlock>>>(d_src, d_dest, numberofelements);

    hipDeviceSynchronize();
    kernel_wct_start = getTimeStamp();
    for (int k=0;k<kernel_reps;k++)
    {
    cudakernel<<<numBlocks, numThreadsPerBlock>>>(d_src, d_dest, numberofelements);
    }
    hipDeviceSynchronize();
    kernel_wct_end = getTimeStamp();

    devtohost_start = getTimeStamp();
    hipMemcpy(a_dest, d_dest, size, hipMemcpyDeviceToHost);
    devtohost_end = getTimeStamp();

    checkOnCPU(a_dest, numberofelements);
    hipFree(a_src);
    hipFree(a_dest);
    hipHostFree(d_src);
    hipHostFree(d_dest);
    /*
    printf("Execution time: %f seconds\n", kernel_wct_end - kernel_wct_start);
    printf("Host to device time: %f seconds\n", hosttodev_end - hosttodev_start);
    printf("Device to host time: %f seconds\n", devtohost_end - devtohost_start);
    */
    double Bw_HosttoDev = size/(hosttodev_end - hosttodev_start);
    double Bw_DevtoHost = size/(devtohost_end - devtohost_start);
    double Bw_execution = (size*2*16)/(kernel_wct_end - kernel_wct_start);
    fprintf(fptr,"%d,",numberofelements);
    fprintf(fptr,"%d,",kernel_reps);
    fprintf(fptr,"%d,",cuda_blocks);
    fprintf(fptr,"%d,",cuda_threads_per_block);
    fprintf(fptr,"%f,",hosttodev_end - hosttodev_start);
    fprintf(fptr,"%f,",kernel_wct_end - kernel_wct_start);
    fprintf(fptr,"%f,",devtohost_end - devtohost_start);
    fprintf(fptr,"%f,",Bw_HosttoDev);
    fprintf(fptr,"%f,",Bw_DevtoHost);
    fprintf(fptr,"%f\n",Bw_execution);
    return 0;
    }
